#include "CudaFunctions.cuh"

__constant__ WindowInfo Info;

struct square
{
	__host__ __device__
		T operator()(const T& x) const
	{
		return x * x;
	}
};


void CudaFunctions::ParticlesMovement(
	Settings & WinSettings,
	std::vector<std::vector<std::vector<Particle>>> & Particles,
	std::vector<std::vector<std::vector<Particle>>> & NextParticles,
	std::vector<std::vector<bool>> & IsFieldOccupied
)
{
	hipMemcpyToSymbol(HIP_SYMBOL(&Info), &WinSettings.Info, sizeof(WindowInfo));

	stde::For_each<NumberOfThreadsRunning>(Particles.begin(), Particles.end(), [&](decltype(Particles.begin()) Iter)
	{
		auto x = std::distance(Particles.begin(), Iter);

		for (int Y = 0; Y < WinSettings.Info.MapY; ++Y)
		{
			for (auto& Item : Iter->data()[Y])
			{
				Item.Move();

				for (int IndexX = stde::Trim(Item.BoundryX.x, 0, WinSettings.Info.MapX); IndexX < Item.BoundryX.y && IndexX < WinSettings.Info.MapX; ++IndexX)
					for (int IndexY = stde::Trim(Item.BoundryY.x, 0, WinSettings.Info.MapY); IndexY < Item.BoundryY.y && IndexY < WinSettings.Info.MapY; ++IndexY)
						for (auto& With : Particles[IndexX][IndexY])
						{
							if (&Item == &With) continue;

							Item.Collide(With);
						}

				Item.Collide(IsFieldOccupied);

				Item.CollideWithBorders();
			}
		}

	});

	stde::For_each<NumberOfThreadsRunning>(Particles.begin(), Particles.end(), [&](decltype(Particles.begin()) Iter)
	{
		auto x = std::distance(Particles.begin(), Iter);
		for (int Y = 0; Y < WinSettings.Info.MapY; ++Y)
		{
			for (auto& Item : Iter->data()[Y])
			{
				Item.Update();

				if (Item.HasLeftMap())
				{
					--WinSettings.ParticlesCounter;
					continue;
				}


				std::lock_guard<std::mutex> _lock(MutArray[stde::Trim(Item.Position.x, 0, WinSettings.Info.MapX - 1)]);
				NextParticles[stde::Trim(Item.Position.x, 0, WinSettings.Info.MapX - 1)][stde::Trim(Item.Position.y, 0, WinSettings.Info.MapY - 1)].push_back(Item);
			}
			Particles[x][Y].clear();
		}
	});


	Particles.swap(NextParticles);

}


